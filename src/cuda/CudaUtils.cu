#include "hip/hip_runtime.h"
#include "pch.h"

#include "clock/Clock.h"
#include "CudaUtils.h"
#include "Utils.h"

#include <hip/hip_runtime.h>
#include <>

namespace ai::cuda
{
    __global__ void FindUniquesKernel(int32_t* array_, uint32_t size) {
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        if (tid < size) {
            array_[tid] += 1;
        }
    }

    std::vector<int32_t> FindUniquesCPU(const std::vector<int32_t>& src, size_t uniqueSize) {
        std::vector<int32_t> result(src.size());

        for (int i = 0; i != src.size(); ++i) {
            result[i] = src[i] + 1;
        }

        return result;
    }


    std::vector<int32_t> FindUniquesGPU(const std::vector<int32_t>& src, size_t uniqueSize) {
        uint32_t size = src.size();
        int32_t* deviceData;

        std::vector<int32_t> result(size);

        Clock clock;
        auto mallocMemcpy = clock.Now();
        hipMalloc(&deviceData, size * sizeof(int32_t));
        hipMemcpy(deviceData, src.data(), size * sizeof(int32_t), hipMemcpyHostToDevice);
        std::cout << "MallocMemcpy: "; clock.PrintDurationFrom(mallocMemcpy);
        
        int totalThreads = size;
        int numThreads = 320;

        auto cudaComp = clock.Now();
        FindUniquesKernel<<<(totalThreads + numThreads - 1) / numThreads, numThreads>>>(deviceData, size);
        std::cout << "CudaComputation: "; clock.PrintDurationFrom(cudaComp);

        auto memcpyFree = clock.Now();
        hipMemcpy(result.data(), deviceData, size * sizeof(int32_t), hipMemcpyDeviceToHost);
        hipFree(deviceData);
        std::cout << "CudaMemcpyFree: "; clock.PrintDurationFrom(memcpyFree);

        return result;
    }
}
